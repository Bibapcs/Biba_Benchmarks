#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 256

__global__ void memory_latency_kernel(float *data, int size) {
    __shared__ float shared_data[BLOCK_SIZE];
    
    // 将数据从全局内存加载到共享内存
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < size) {
        shared_data[threadIdx.x] = data[idx];
    }
    
    // 同步线程块
    __syncthreads();
    
    // 从共享内存中读取数据
    if (idx < size) {
        data[idx] = shared_data[threadIdx.x];
    }
}

void measure_memory_latency(int block_size_kb) {
    int block_size = block_size_kb * 1024; // 将KB转换为字节
    float *h_data, *d_data;
    
    // 分配主机和设备内存
    h_data = (float *)malloc(block_size);
    hipMalloc((void **)&d_data, block_size);
    
    // 初始化数据
    for (int i = 0; i < block_size / sizeof(float); ++i) {
        h_data[i] = (float)i;
    }
    
    // 复制数据到设备
    hipMemcpy(d_data, h_data, block_size, hipMemcpyHostToDevice);
    
    // 创建CUDA事件
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // 启动计时器
    hipEventRecord(start);
    
    // 执行核函数
    int num_blocks = (block_size / sizeof(float) + BLOCK_SIZE - 1) / BLOCK_SIZE;
    memory_latency_kernel<<<num_blocks, BLOCK_SIZE>>>(d_data, block_size / sizeof(float));
    
    // 停止计时器
    hipEventRecord(stop);
    hipDeviceSynchronize();
    
    float elapsed_time = 0.0f;
    hipEventElapsedTime(&elapsed_time, start, stop);
    
    // 输出结果
    printf("Block size: %d KB, Latency: %.6f ms\n", block_size_kb, elapsed_time);
    
    // 释放资源
    hipEventDestroy(start);
    hipEventDestroy(stop);
    free(h_data);
    hipFree(d_data);
}

int main() {
    // 初始化CUDA环境
    hipSetDevice(0); // 设置GPU设备
    
    printf("Measuring memory latency for different block sizes...\n");

    // warm up
    measure_memory_latency(1);
    printf("Warm up over, start bench...\n");
    
    // 测量不同块大小的延迟
    for (int block_size_kb = 1; block_size_kb <= 1048576; block_size_kb *= 2) {
        measure_memory_latency(block_size_kb);
    }
    
    return 0;
}
